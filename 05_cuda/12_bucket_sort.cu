
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>


__global__ void bucketSort(int* key, int * bucket, int n, int range){

    //__deice__ __managed__ int
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    bucket[i] = 0;
    //extern __shared__ std::vector<int> bucket(range);
    //for(int j=1; j<range; j++){
    //    bucket[j] = 0;
    //}



    for(int j=0; j<n; j++){
        if(key[j]==i){
            bucket[i]++;
        }
    }

    __syncthreads();
    int k = 0;
    for(int t = 0;t<i;t++){
        k+=bucket[t];
    }


    for(;bucket[i]>0;bucket[i]--){
        key[k++]=i;

    }

    __syncthreads();
};
int main() {
  int n = 50;
  int range = 5;
  //std::vector<int> key(n); no stl in device
  //std::vector<int> bucket(range);
  int  *key,*bucket;

  hipMallocManaged(&key, n*sizeof(int));
  hipMallocManaged(&bucket, range*sizeof(int));

  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");



  bucketSort<<<1,range>>>(key, bucket, n,range);
  hipDeviceSynchronize();

/*
  for (int i=0; i<range; i++) {
    bucket[i] = 0;
  }
  for (int i=0; i<n; i++) {
    bucket[key[i]]++;
  }

  for (int i=0, j=0; i<range; i++) {
    for (; bucket[i]>0; bucket[i]--) {
      key[j++] = i;
    }
  }
  */




  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
  hipFree(bucket);
  hipFree(key);
}
